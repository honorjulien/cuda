#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "lodepng.h"
#include <hip/hip_runtime.h>
 
#define SIGMA_S 2.0
#define SIGMA_R 50.0
#define KERNEL_SIZE 5
 
typedef struct {
    unsigned char *image;
    unsigned width, height;
} Image;
 
void load_image(const char *filename, Image *img) {
    unsigned error = lodepng_decode32_file(&img->image, &img->width, &img->height, filename);
    if (error) {
        printf("Erreur lors du chargement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}
 
void save_image(const char *filename, Image *img) {
    unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);
    if (error) {
        printf("Erreur lors de l'enregistrement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}
 
__global__ void bilateral_filter_cuda(unsigned char *d_input, unsigned char *d_output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
 
    if (x >= width || y >= height) return;
 
    int idx = 4 * (y * width + x);
    float sum_r = 0, sum_g = 0, sum_b = 0, norm_factor = 0;
    float sigma_s2 = 2.0 * SIGMA_S * SIGMA_S;
    float sigma_r2 = 2.0 * SIGMA_R * SIGMA_R;
    int half_size = KERNEL_SIZE / 2;
 
    unsigned char r = d_input[idx], g = d_input[idx+1], b = d_input[idx+2];
 
    for (int i = -half_size; i <= half_size; i++) {
        for (int j = -half_size; j <= half_size; j++) {
            int yy = y + i;
            int xx = x + j;
            if (xx >= 0 && xx < width && yy >= 0 && yy < height) {
                int neighbor_idx = 4 * (yy * width + xx);
                unsigned char nr = d_input[neighbor_idx], ng = d_input[neighbor_idx+1], nb = d_input[neighbor_idx+2];
                float intensity_diff = (r - nr) * (r - nr) + (g - ng) * (g - ng) + (b - nb) * (b - nb);
                float range_weight = expf(-intensity_diff / sigma_r2);
                float spatial_weight = expf(-(i * i + j * j) / sigma_s2);
                float weight = spatial_weight * range_weight;
                sum_r += weight * nr;
                sum_g += weight * ng;
                sum_b += weight * nb;
                norm_factor += weight;
            }
        }
    }
 
    d_output[idx] = (unsigned char)(sum_r / norm_factor);
    d_output[idx+1] = (unsigned char)(sum_g / norm_factor);
    d_output[idx+2] = (unsigned char)(sum_b / norm_factor);
    d_output[idx+3] = d_input[idx+3];
}
 
void bilateral_filter(Image *img) {
    int width = img->width;
    int height = img->height;
    size_t img_size = width * height * 4 * sizeof(unsigned char);
    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMemcpy(d_input, img->image, img_size, hipMemcpyHostToDevice);
 
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    bilateral_filter_cuda<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    hipMemcpy(img->image, d_output, img_size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}
 
int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s input.png output.png\n", argv[0]);
        return 1;
    }
 
    Image img;
    load_image(argv[1], &img);
    bilateral_filter(&img);
    save_image(argv[2], &img);
 
    free(img.image);
    return 0;
}