#include "hip/hip_runtime.h"
// ======================================================================================
// Gestion des librairies et des variables
// ======================================================================================
// Import des librairies
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>  // Pour mesurer le temps d'exécution
#include "lodepng.h"
#include <hip/hip_runtime.h>
#include <time.h>  // Pour mesurer le temps d'exécution

// Definition des variables
#define SIGMA_S 2.0   // Paramètre pour la composante spatiale
#define SIGMA_R 50.0  // Paramètre pour la composante de similitude d'intensité
#define KERNEL_SIZE 5 // Taille du noyau pour le filtrage

// Structure pour stocker les données d'une image
typedef struct {
    unsigned char *image; // Pointeur vers les pixels de l'image
    unsigned width, height; // Dimensions de l'image
} Image;


// ======================================================================================
// Chargement et sauvegarde de l'image
// ======================================================================================
// Fonction pour charger une image PNG à partir d'un fichier
void load_image(const char *filename, Image *img) {
    unsigned error = lodepng_decode32_file(&img->image, &img->width, &img->height, filename);
    if (error) {
        printf("Erreur lors du chargement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}

// Fonction pour enregistrer une image PNG dans un fichier
void save_image(const char *filename, Image *img) {
    unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);
    if (error) {
        printf("Erreur lors de l'enregistrement de l'image: %s\n", lodepng_error_text(error));
        exit(1);
    }
}


// ======================================================================================
// Filtre bilatéral en CUDA
// ======================================================================================
// Fonction CUDA pour appliquer le filtre bilatéral
__global__ void bilateral_filter_cuda(unsigned char *d_input, unsigned char *d_output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
 
    if (x >= width || y >= height) return; // Si le thread est en dehors des limites de l'image, on quitte

    int idx = 4 * (y * width + x); // Index du pixel dans l'image (r, g, b, a)
    float sum_r = 0, sum_g = 0, sum_b = 0, norm_factor = 0;
    float sigma_s2 = 2.0 * SIGMA_S * SIGMA_S;
    float sigma_r2 = 2.0 * SIGMA_R * SIGMA_R;
    int half_size = KERNEL_SIZE / 2;
 
    unsigned char r = d_input[idx], g = d_input[idx+1], b = d_input[idx+2];
 
    // Calcul du filtre bilatéral pour chaque pixel en utilisant ses voisins
    for (int i = -half_size; i <= half_size; i++) {
        for (int j = -half_size; j <= half_size; j++) {
            int yy = y + i;
            int xx = x + j;
            if (xx >= 0 && xx < width && yy >= 0 && yy < height) { // Vérifie si le voisin est dans l'image
                int neighbor_idx = 4 * (yy * width + xx);
                unsigned char nr = d_input[neighbor_idx], ng = d_input[neighbor_idx+1], nb = d_input[neighbor_idx+2];
                float intensity_diff = (r - nr) * (r - nr) + (g - ng) * (g - ng) + (b - nb) * (b - nb);
                float range_weight = expf(-intensity_diff / sigma_r2);  // Poids basé sur la différence d'intensité
                float spatial_weight = expf(-(i * i + j * j) / sigma_s2);  // Poids spatial
                float weight = spatial_weight * range_weight;
                sum_r += weight * nr;
                sum_g += weight * ng;
                sum_b += weight * nb;
                norm_factor += weight;
            }
        }
    }
 
    d_output[idx] = (unsigned char)(sum_r / norm_factor);
    d_output[idx+1] = (unsigned char)(sum_g / norm_factor);
    d_output[idx+2] = (unsigned char)(sum_b / norm_factor);
    d_output[idx+3] = d_input[idx+3]; // Conserve la composante alpha
}

// Fonction pour appliquer le filtre bilatéral avec CUDA
void bilateral_filter(Image *img) {
    int width = img->width;
    int height = img->height;
    size_t img_size = width * height * 4 * sizeof(unsigned char);
    unsigned char *d_input, *d_output;
    
    // Allocation de la mémoire sur le GPU
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMemcpy(d_input, img->image, img_size, hipMemcpyHostToDevice);
 
    // Dimensions des blocs et de la grille CUDA
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    
    // Lancement du noyau CUDA pour appliquer le filtre bilatéral
    bilateral_filter_cuda<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    
    // Copie des résultats du GPU vers le CPU
    hipMemcpy(img->image, d_output, img_size, hipMemcpyDeviceToHost);
    
    // Libération de la mémoire GPU
    hipFree(d_input);
    hipFree(d_output);
}


// ======================================================================================
// Fonction main pour exécuter le programme
// ======================================================================================
int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s input.png output.png\n", argv[0]);
        return 1;
    }

    Image img;

    // Chargement de l'image
    load_image(argv[1], &img);

    // Démarrage du chronomètre
    clock_t start_time = clock();

    // Application du filtre bilatéral
    bilateral_filter(&img);

    // Arrêt du chronomètre
    clock_t end_time = clock();

    // Calcul du temps d'exécution en secondes
    double total_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    // Sauvegarde de l'image résultante
    save_image(argv[2], &img);

    // Affichage du temps d'exécution
    printf("Temps d'exécution : %.4f secondes\n", total_time);

    // Libération de la mémoire de l'image
    free(img.image);

    return 0;
}
